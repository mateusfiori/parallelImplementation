#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <windows.h>
#include <conio.h>
#include <time.h>


__global__ void preencheCentroid(float avgDiss[][2], float grupoG[], float centroid[], int qteElementos)
{
	int i = threadIdx.x;

	if (i < qteElementos) {
		avgDiss[i][0] = sqrt(((grupoG[i * 3] - centroid[i * 3]) * (grupoG[i * 3] - centroid[i * 3])) + ((grupoG[i * 3 + 1] - centroid[i * 3 + 1]) * (grupoG[i * 3 + 1] - centroid[i * 3 + 1])));
		avgDiss[i][1] = grupoG[i * 3 + 2];
	}
}


//CODIGO EM C
//*************************************************************************************************************************************//

//Grupos de elementos(objetos) ser�o identificados por n�meros come�ando do 0 (ZERO)
//Portanto o primeiro grupo ser� o grupo 0 
//A estrutura funciona da seguinte forma: float estrutura[grupo][elementos do grupo][informa��o de cada elemento -> 0 => x, 1 => y, 2 => index]

//para aumentar o numero de grupo � preciso alterar as declara��es: maxGrupo e cuboDeDados e tambem a fun��o que preenche o cubo

#define NUM_GRUPOS 90
#define NUM_MAX_ELEMENTOS 100

void preencheEstrutura(float cuboDeDados[][100][3]) {

	for (int i = 0; i < 100; i++)
		for (int j = 0; j < 100; j++)
			for (int k = 0; k < 3; k++)
				cuboDeDados[i][j][k] = -1.f;

}

void insereDadosExternos(float cuboDeDados[][100][3], int dadosExternos[][2]) {

	int index = 0; //identifica��o �nica dos elementos come�a no 0 (zero) 	

	for (int i = 0; i < 90; i++) {

		for (int j = 0; j < 2; j++)
			cuboDeDados[0][i][j] = dadosExternos[i][j];

		cuboDeDados[0][i][2] = index++;
	}
} //fim da fun��o

void coletaDadosExternos(int dadosExternos[][2]) {

	FILE *arqExterno;

	arqExterno = fopen("C:\\Users\\cliente\\Desktop\\cudaProjects\\centroidParalelo\\centroidParalelo\\duun.txt", "r"); //arquivo cedido pela Hethini (90 pontos)

										 //rotina de erro de abertura de arquivo
	if (arqExterno == NULL)
	{
		printf("Error Reading File\n");
		exit(0);
	}

	for (int i = 0; i < 90; i++)
		for (int j = 0; j < 2; j++)
			fscanf(arqExterno, "%d ", &dadosExternos[i][j]);

} //fim da fun��o

void mostraElementosDoGrupo(float cuboDeDados[][100][3], int grupo) {

	printf("X [0]\tY [1]\tINDEX [2]\n\n"); //cabe�alho dos elementos
	for (int i = 0; i < NUM_MAX_ELEMENTOS; i++) {

		for (int j = 0; j < 3; j++) {

			if (cuboDeDados[grupo][i][j] >= 0)
				printf("%.2f\t", cuboDeDados[grupo][i][j]); //print de todos os elementos do grupo desejado
		}

		if (cuboDeDados[grupo][i][0] >= 0 && cuboDeDados[grupo][i][1] >= 0 && cuboDeDados[grupo][i][2] >= 0)
			printf("\n");

	}

} //fim da fun��o

void insereDadosG(float grupoG[][3], int dadosExternos[][2]) {

	int index = 0;

	for (int i = 0; i < 90; i++) {

		for (int j = 0; j < 2; j++)
			grupoG[i][j] = dadosExternos[i][j];

		grupoG[i][2] = index++;
	}

}

void mostraMatrix(float matrix[][3]) {


	printf("X [0]\tY [1]\tINDEX [2]\n\n"); //cabe�alho dos elementos
	for (int i = 0; i < 90; i++) {

		for (int j = 0; j < 3; j++)
			printf("%.2f\t", matrix[i][j]); //print de todos os elementos de uma matrix

		printf("\n");

	}

}

void mostraMatrixDx(float matrix[][2]) {


	printf("Media [0]\tINDEX [1]\n\n"); //cabe�alho dos elementos
	for (int i = 0; i < 90; i++) {

		for (int j = 0; j < 2; j++)
			printf("%.2f\t\t", matrix[i][j]); //print de todos os elementos de uma matrix

		printf("\n");

	}

}

void inicializaMatrizDiss(float matrizDissimilaridade[][91]) {

	for (int i = 0; i < 91; i++)
		for (int j = 0; j < 91; j++)
			matrizDissimilaridade[i][j] = -1;

}

void preencheMatrizDiss(float matrizDissimilaridade[][91], float grupoG[][3], int qteElementos) {

	//montar uma matriz de dissimilaridade
	//matrizes de dissimilaridade s�o preenchidas com dist�ncias euclidianas

	for (int i = 0; i < qteElementos; i++) {

		for (int j = 0; j < qteElementos; j++)
			matrizDissimilaridade[i][j] = sqrt((grupoG[i][0] - grupoG[j][0]) * (grupoG[i][0] - grupoG[j][0]) +
			(grupoG[i][1] - grupoG[j][1]) * (grupoG[i][1] - grupoG[j][1])); // distancia euclidiana

		matrizDissimilaridade[i][qteElementos] = grupoG[i][2]; //ultima posi��o � reservada para a identifica��o do elemento
	}
}

void mostraMatrixDiss(float matrix[][91]) {


	printf("X [0]\tY [1]\tINDEX [2]\n\n"); //cabe�alho dos elementos
	for (int i = 0; i < 50; i++) {

		for (int j = 0; j < 15; j++)
			printf("%.2f\t", matrix[i][j]); //print de todos os elementos da matrix

		printf("\n");

	}

}

void preencheMatrizAVG(float matrizDissimilaridade[][91], float avgDiss[][2], int qteElementos) {

	float soma = 0.0;

	for (int i = 0; i < qteElementos; i++) {

		for (int j = 0; j < qteElementos; j++)
			soma += matrizDissimilaridade[i][j];

		avgDiss[i][0] = soma;
		avgDiss[i][1] = matrizDissimilaridade[i][qteElementos];
		soma = 0.0;
	}

}

void mostraMatrixAVG(float matrix[][2]) {


	printf("Media [0]\tINDEX [1]\n\n"); //cabe�alho dos elementos
	for (int i = 0; i < 90; i++) {

		for (int j = 0; j < 2; j++)
			printf("%f\t\t", matrix[i][j]); //print de todos os elementos de uma matrix

		printf("\n");

	}

}

float identificaMaiorDiss(float avgDiss[][2], int qteElementos) {

	float maiorDiss = avgDiss[0][0];
	float id = avgDiss[0][1];

	for (int i = 0; i < qteElementos; i++) {
		if (avgDiss[i][0] > maiorDiss) {
			maiorDiss = avgDiss[i][0];
			id = avgDiss[i][1];
		}
	}

	return id;
}

float identificaMaiorDx(float Dx[][2], int qteElementos) {

	float maiorDx, id;

	maiorDx = Dx[0][0];
	id = Dx[0][1];

	for (int i = 0; i < qteElementos; i++) {

		if (Dx[i][0] > maiorDx) {
			maiorDx = Dx[i][0];
			id = Dx[i][1];
		}
	}

	if (maiorDx <= 0.0) return -1; //retorna negativo se o maior valor for negativo e portanto para o loop
	else	return id;

}

void inicializaMatrizNegativo(float matrizG[][3]) {

	for (int i = 0; i < 90; i++)
		for (int j = 0; j < 3; j++)
			matrizG[i][j] = -1;

}

//inicializa a matriz Dx com -99999.99, numero que � improvavel de acontecer
void inicializaMatrizDx(float Dx[][2]) {

	for (int i = 0; i < 90; i++)
		for (int j = 0; j < 2; j++)
			Dx[i][j] = -99999.99;

}

void deletaElementoDiss(float grupoG[][3], float elementoMaisDissimilar, int *qteElementos, float elementoAux[]) {

	for (int i = 0; i < *qteElementos; i++) {

		if (grupoG[i][2] == elementoMaisDissimilar) {

			//salva em uma estrutura o elemento
			elementoAux[0] = grupoG[i][0];
			elementoAux[1] = grupoG[i][1];
			elementoAux[2] = grupoG[i][2];

			//deleta o elemento com o index desejado do grupo
			for (int j = i; j < *qteElementos - 1; j++) {
				grupoG[j][0] = grupoG[j + 1][0];
				grupoG[j][1] = grupoG[j + 1][1];
				grupoG[j][2] = grupoG[j + 1][2];
			}

			//deleta o ultimo elemento	
			grupoG[*qteElementos - 1][0] = -1.0; //coloca -1 na ultima posi��o e depois diminui a quantidade de elementos	
			grupoG[*qteElementos - 1][1] = -1.0;
			grupoG[*qteElementos - 1][2] = -1.0; //coloca -1 na ultima posi��o e depois diminui a quantidade de elementos
			*qteElementos -= 1;
			break;
		}

	}//fim do for de delete

}

void colocaElementoTempG(float tempG[][3], float elementoAux[]) {

	for (int i = 0; i < 90; i++)
		if (tempG[i][0] < 0) {

			tempG[i][0] = elementoAux[0];
			tempG[i][1] = elementoAux[1];
			tempG[i][2] = elementoAux[2];
			break;
		}

}

//partindo do principio que o max de elementos � 90, quando aumentar a quantidade de dados tem que mudar essa funcao
int contaElementosTempG(float tempG[][3]) {

	int quantidade = 0;

	for (int i = 0; i < 90; i++) {

		if (tempG[i][0] < 0)
			break;
		else {
			quantidade++;
		}

	}

	return quantidade;
}

int contaElementosGrupoG(float grupoG[][3]) {

	int quantidade = 0;

	for (int i = 0; i < 90; i++) {

		if (grupoG[i][0] < 0)
			break;
		else {
			quantidade++;
		}

	}

	return quantidade;
}

void preencheDx(float Dx[][2], float grupoG[][3], float tempG[][3], int qteElementos, int qteElementosTempG) {

	float somaG, distG, somaTempG, distTempG;
	int cont;

	somaTempG = 0;
	somaG = 0;
	distTempG = 0;
	distG = 0;
	cont = 0;

	for (int i = 0; i < qteElementos; i++) {

		somaTempG = 0;
		somaG = 0;

		//esse for ira somar as distancias de i com rela��o a todos os elementos do grupoG
		for (int j = 0; j < qteElementos; j++) {

			if (tempG[j][0] >= 0.f) {

				cont++;
				distTempG = sqrt((grupoG[i][0] - tempG[j][0]) * (grupoG[i][0] - tempG[j][0]) +
					(grupoG[i][1] - tempG[j][1]) * (grupoG[i][1] - tempG[j][1]));
				somaTempG += distTempG;
			} //fim do if

			distG = sqrt((grupoG[i][0] - grupoG[j][0]) * (grupoG[i][0] - grupoG[j][0]) +
				(grupoG[i][1] - grupoG[j][1]) * (grupoG[i][1] - grupoG[j][1]));
			somaG += distG;

		} //fim do segundo for

		  //atribui��o da diferen�a da soma a estrutura D(x)
		somaG /= (qteElementos - 1); // qteElementos-1 pois n�o se leva em considera��o a distancia do elemento de g com rela��o a ele mesmo
		somaTempG /= qteElementosTempG;

		Dx[i][0] = somaG - somaTempG; //diferen�a das distancias
		Dx[i][1] = grupoG[i][2]; //index do elemento

	} //fim do primeiro for

	printf("\nCONT COM QTETEMPG: %d\n", cont);
}

void preencheCuboComG(float cuboDeDados[][NUM_MAX_ELEMENTOS][3], float grupoG[][3], int indexMaiorDiametro, int qteElementos) {

	for (int i = 0; i < NUM_MAX_ELEMENTOS; i++) {

		if (i < qteElementos) {

			cuboDeDados[indexMaiorDiametro][i][0] = grupoG[i][0];
			cuboDeDados[indexMaiorDiametro][i][1] = grupoG[i][1];
			cuboDeDados[indexMaiorDiametro][i][2] = grupoG[i][2];

		}
		else {

			cuboDeDados[indexMaiorDiametro][i][0] = -1;
			cuboDeDados[indexMaiorDiametro][i][1] = -1;
			cuboDeDados[indexMaiorDiametro][i][2] = -1;

		}

	}

}

int encontraGrupoVazio(float cuboDeDados[][NUM_MAX_ELEMENTOS][3]) {

	int grupoVazio = -1;

	for (int i = 0; i < 100; i++) {

		if (cuboDeDados[i][0][0] < 0.f) {
			grupoVazio = i;
			break;
		}

	}

	return grupoVazio;

}

void preencheCuboComTempG(float cuboDeDados[][NUM_MAX_ELEMENTOS][3], float tempG[][3], int grupoVazio, int qteElementosTempG) {

	for (int i = 0; i < NUM_MAX_ELEMENTOS; i++) {

		if (i < qteElementosTempG) {

			cuboDeDados[grupoVazio][i][0] = tempG[i][0];
			cuboDeDados[grupoVazio][i][1] = tempG[i][1];
			cuboDeDados[grupoVazio][i][2] = tempG[i][2];

		}
		else {

			cuboDeDados[grupoVazio][i][0] = -1;
			cuboDeDados[grupoVazio][i][1] = -1;
			cuboDeDados[grupoVazio][i][2] = -1;

		}

	}

}

void inicializaMatrizDiametro(float diametroDoGrupo[][2]) {

	for (int i = 0; i < NUM_GRUPOS; i++) {

		diametroDoGrupo[i][0] = -1.f;
		diametroDoGrupo[i][1] = -1.f;

	}
}

int contaElementosCubo(float cuboDeDados[][NUM_MAX_ELEMENTOS][3], int grupo) {

	int cont = 0;

	for (int i = 0; i < NUM_MAX_ELEMENTOS; i++)
		if (cuboDeDados[grupo][i][0] < 0) break;
		else cont++;
		return cont;
}

void preencheMatrizDeDiametro(float cuboDeDados[][NUM_MAX_ELEMENTOS][3], float diametroDoGrupo[][2]) {

	float distanciaParcial, distanciaFinal;
	int quantidadeDeElementosNoGrupo, k;

	k = 0;

	do {

		quantidadeDeElementosNoGrupo = contaElementosCubo(cuboDeDados, k);
		distanciaFinal = 0;

		for (int i = 0; i < 100; i++) {

			distanciaParcial = 0;

			if (cuboDeDados[k][i][0] < 0) break;
			else {

				for (int j = 0; j < 100; j++) {


					if (cuboDeDados[k][j][0] < 0) break;
					else {

						distanciaParcial += sqrt((cuboDeDados[k][i][0] - cuboDeDados[k][j][0]) * (cuboDeDados[k][i][0] - cuboDeDados[k][j][0]) +
							(cuboDeDados[k][i][1] - cuboDeDados[k][j][1]) * (cuboDeDados[k][i][1] - cuboDeDados[k][j][1]));
					}
				}
			}

			distanciaFinal += distanciaParcial;
		}

		//se tiver s� um elemento no grupo o diametro � zero
		if (quantidadeDeElementosNoGrupo > 1)
			diametroDoGrupo[k][0] = distanciaFinal / (quantidadeDeElementosNoGrupo*(quantidadeDeElementosNoGrupo - 1));
		else diametroDoGrupo[k][0] = 0.f;

		diametroDoGrupo[k][1] = k;


		k++;
	} while (cuboDeDados[k][0][0] >= 0);

}

float identificaMaiorDiametro(float diametroDoGrupo[][2]) {

	float maiorDiametro = diametroDoGrupo[0][0];
	float id = diametroDoGrupo[0][1];

	for (int i = 0; i < NUM_GRUPOS; i++) {
		if (diametroDoGrupo[i][0] > maiorDiametro) {
			maiorDiametro = diametroDoGrupo[i][0];
			id = diametroDoGrupo[i][1];
		}
	}

	return id;
}

void deletaElementoDx(float Dx[][2], int qteDx, float idDx) {

	for (int i = 0; i < qteDx; i++) {

		if (Dx[i][1] == idDx) {

			//deleta o elemento de Dx
			for (int j = i; j < qteDx - 1; j++) {
				Dx[j][0] = Dx[j + 1][0];
				Dx[j][1] = Dx[j + 1][1];
			}

			Dx[qteDx - 1][0] = -99999.99;
			Dx[qteDx - 1][1] = -99999.99;
			break;

		}


	}

}

int contaElementosDx(float Dx[][2]) {

	int quantidade = 0;

	for (int i = 0; i < 90; i++) {

		if (Dx[i][1] < 0)
			break;
		else {
			quantidade++;
		}

	}

	return quantidade;
}

void preencheGcomMaiorDiametro(float cuboDeDados[][NUM_MAX_ELEMENTOS][3], float grupoG[][3], int indexMaiorDiametro) {

	for (int i = 0; i < 100; i++)
		for (int j = 0; j < 3; j++)
			grupoG[i][j] = cuboDeDados[indexMaiorDiametro][i][j];

}

void calculaCentroid(float grupoG[][3], float centroid[][3], int qteElementos) {

	float centroidGERAL[2], somaX = 0, somaY = 0;

	for (int i = 0; i < qteElementos; i++) {
		somaX += grupoG[i][0];
		somaY += grupoG[i][1];
	}

	centroidGERAL[0] = somaX / qteElementos;
	centroidGERAL[1] = somaY / qteElementos;

	for (int i = 0; i < qteElementos; i++) {
		centroid[i][0] = centroidGERAL[0] - grupoG[i][0];
		centroid[i][1] = centroidGERAL[1] - grupoG[i][1];
		centroid[i][2] = grupoG[i][2];
	}
}

void caculaDistCentroid(float avgDiss[][2], float centroid[][3], float grupoG[][3], int qteElementos) {

	for (int i = 0; i < qteElementos; i++) {

		avgDiss[i][0] = sqrt(((grupoG[i][0] - centroid[i][0]) * (grupoG[i][0] - centroid[i][0])) + ((grupoG[i][1] - centroid[i][1]) * (grupoG[i][1] - centroid[i][1])));
		avgDiss[i][1] = grupoG[i][2];
	}

}

int main() {

	float start, delta;
	float grupoG_array[270], centroid_array[270], (*d_avgDiss)[2], *d_grupoG, *d_centroid;
	float cuboDeDados[100][NUM_MAX_ELEMENTOS][3]; //estrutura principal, x, y e index de cada elemento de cada grupo (AGt)
	float matrizDissimilaridade[91][91]; //est�tica por�m pode ser implementada din�mica e possui 90+1 espa�os pois o ultimo espa�o � reservado para o index do elemento
	float grupoG[90][3]; //grupo a ser trabalhado no la�o do algoritmo (G) => (x, y, index)
	float tempG[90][3]; //grupo que armazena os elementos que n�o fazem parte do grupoG
	float grupoTempG[90][3]; //grupo auxiliar utilizado para divis�o de grupos
	float avgDiss[90][2]; //matriz que armazena as medias de dissimilaridade de cada elemento
	float maiorDiss[2]; //estrutura que armazena a maior distancia e qual elemento ela pertence
	float elementoAux[3];//estrutura que guarda o elemento que sera deletado e colocado em tempG
	float Dx[90][2]; //matriz que armazenara a diferen�a das somas das distancias e o index do elemento
	float idDx;
	float diametroDoGrupo[NUM_GRUPOS][2]; //matriz que armazena o diametro e o grupo que ele pertence
	int dadosExternos[90][2]; // matriz que armazena os dados vindo externamente (arquivo .txt) (X)
	int maxGrupos, it, qteElementos, qteElementosTempG, maxIt, indexMaiorDiametro, qteDx;
	float centroid[90][3];

	//defini��es de vari�veis
	maxGrupos = NUM_GRUPOS; //numero maximo de grupos
	it = 0; //numero de itera��es
	qteElementos = 90; //quantidade de elementos existentes no grupoG
	qteElementosTempG = 0; //valor inicial, sera mudado posteriormente pela fun��o contaElementosTempG
	idDx = 0; //inicia com algum valor o id que sera retornado em Dx
	maxIt = 0; //numero de itera��es do segundo la�o, usado para sair do la�o caso atinja o numero maximo
	indexMaiorDiametro = 0; //variavel que armazena o index do grupo da estrutura principal com o maior diametro, para posteriormente ser substituido pelo grupoG
							//defaul � 0 (zero) pois todos os dados externos s�o postos no grupo 0 (zero)
	qteDx = 0; // inicia a quantidade de elementos em Dx

	start = GetTickCount();

	//Preenche as caracteristicas de cada dado com um valor especificado (-1)
	preencheEstrutura(cuboDeDados);

	//pega os dados de um arquivo e coloca na matriz dadosExternos (X), matriz que vai ser posta na estrutura posteriormente	
	coletaDadosExternos(dadosExternos); //(X)

	//insere os dodos externos na estrutura, todos os elementos est�o em um s� grupo
	insereDadosExternos(cuboDeDados, dadosExternos); // ** (AGt <- X) **

	//insere os dados externos (X) no grupo G (grupoG)
	insereDadosG(grupoG, dadosExternos); // **(G <- X)**

	//mostra elementos da matrix (grupoG)
	//mostraMatrix(grupoG);

	//mostra na tela os elementos de um grupo em especifico
	//mostraElementosDoGrupo(cuboDeDados, 0); 

	//****at� este momento os dados est�o todos agrupados em um s� grupo***//

	//Aqui come�a o la�o principal (do while)

	do {

		printf("\n\nITERACAO %d\n\n", it);

		maxIt = 0;

		//mostraMatrix(grupoG);

		//inicia matriz centroid
		inicializaMatrizNegativo(centroid);

		//calcula a centroid a qual o elemento tem que ser calculado
		calculaCentroid(grupoG, centroid, qteElementos);

		inicializaMatrizDx(avgDiss);

		//rotina para o paralelismo
		//transforma grupoG e centroid em vetor
		int contador = 0;
		for (int i = 0; i < 90; i++)
			for (int j = 0; j < 3; j++) {
				grupoG_array[contador] = grupoG[i][j];
				centroid_array[contador] = centroid[i][j];
				contador++;
			}

		//aloca memoria no device
		hipMalloc((float**)&d_avgDiss, 90 * 2 * sizeof(float));
		hipMalloc((float**)&d_grupoG, 90 * 3 * sizeof(float));
		hipMalloc((float**)&d_centroid, 90 * 3 * sizeof(float));

		//passa o vetor para o device com cudamemcpy
		hipMemcpy(d_grupoG, grupoG_array, 90 * 3 * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_centroid, centroid_array, 90 * 3 * sizeof(float), hipMemcpyHostToDevice);

		dim3 block(90);
		dim3 grid(1);

		preencheCentroid << <grid, block >> > (d_avgDiss, d_grupoG, d_centroid, qteElementos);
		hipDeviceSynchronize();

		hipMemcpy(avgDiss, d_avgDiss, (90 * 2) * sizeof(float), hipMemcpyDeviceToHost);

		// calcula as distancias euclidianas
		//caculaDistCentroid(avgDiss, centroid, grupoG, qteElementos); // CENTROID

		//� necess�rio uma matriz com essa estrutura (dist, index)
		//preenche a matriz com as m�dias de distancia de cada elemento
		//preencheMatrizAVG(matrizDissimilaridade, avgDiss, qteElementos); //average linkage -> tera que trocar por centroid linkage	

		//mostra a matrix de medias de distancias
		//mostraMatrixAVG(avgDiss);
		//getch();

																	 //fun��o para encontrar o elemento com a maior media de distancias	
		printf("\nMaior dissimilaridade: %d <- INDEX\n\n", (int)identificaMaiorDiss(avgDiss, qteElementos));

		//inicializa tempG com valores negativos
		inicializaMatrizNegativo(tempG); //todos as posi��es de tempG est�o preenchidas com numeros negativos

										 //identifica o elemento de maior dissimilaridade, o deleta do grupoG e salva o elemento em elementoAux para depois ser posto no grupo tempG
		deletaElementoDiss(grupoG, identificaMaiorDiss(avgDiss, qteElementos), &qteElementos, elementoAux);
		//coloca o elemento em tempG 
		colocaElementoTempG(tempG, elementoAux);

		//preenche as posi�es com -99999,99
		inicializaMatrizDx(Dx);

		//conta quantos elementos tem em tempG
		qteElementosTempG = contaElementosTempG(tempG);

		//printf("\nQTE G: %d   QTE tempG: %d\n", qteElementos, qteElementosTempG);

		//matriz que cont�m a diferen�a das m�dias das distancias de cada elemento do grupoG 
		//com rela��o aos elementos do grupoG menos as medias das distancias do elementos em rela��o aos elementos do grupo tempG
		preencheDx(Dx, grupoG, tempG, qteElementos, qteElementosTempG); //Dx -> (diferen�a das medias, index)	

		printf("\nmax it%d\n", maxIt);
		mostraMatrixAVG(Dx);


		//a partir de agora come�a o segundo la�o que rodara ate que n�o existam mais valores Dx positivos	
		do {


			maxIt++;

			//conta quantos elementos tem em Dx
			qteDx = contaElementosDx(Dx);


			//conta quantos elementos tem em tempG
			qteElementosTempG = contaElementosTempG(tempG);

			//o elemento que obtiver maior D(x) sera tirado do grupoG e colocado no grupo tempG
			//identifica o maior Dx, se o valor do index for negativo � porque todos os valores de Dx s�o negativos, portanto deve-se sair do la�o
			idDx = identificaMaiorDx(Dx, qteElementos);
			printf("\nMaior elemento de Dx: %f <- INDEX\n\n", idDx);


			//checa se o maior valor de Dx � negativo
			//esse if estara dentro de um la�o e quando o valor de idDx for negativo tem que sair do la�o
			if (idDx >= 0 && qteElementos > 1) {

				//deleta o elemento de Dx
				deletaElementoDx(Dx, qteDx, idDx);

				printf("EU SOU NUMERO QUATRO");
				//proximo passo � retirar o elemento encontrado do grupoG e coloca-lo em tempG
				//deleta elemento do grupoG
				deletaElementoDiss(grupoG, idDx, &qteElementos, elementoAux);
				//adiciona elemento em tempG
				colocaElementoTempG(tempG, elementoAux);
			}

			//if (it >= 2) getch(); 
			//else 
			//Sleep(500);

			//intru��o para o la�o nunca cair em loop infinito
			if (maxIt == 100) printf("\n\nNumero maximo de iteracoes atingido.");

		} while (idDx >= 0 && maxIt < 100);

		printf("\nMAXIT TOTAL %d\n", maxIt);


		//todas as variaveis precisam ser atualizadas
		it++;
		printf("\n\nIteracao %d\n\n", it);

		//os grupos precisam ser postos na estrutura principal
		//indexMaiorDiametro � grupo da estrutura principal que deve ser posto o grupoG

		//grupoG � posto na estrutura principal
		preencheCuboComG(cuboDeDados, grupoG, indexMaiorDiametro, qteElementos);

		//procura por um grupo vazio na estrutura principal para se colocar o grupo tempG
		//se retornar um valor negativo � pq nao existem grupos vazios
		printf("\n\nGrupo vazio: %d", encontraGrupoVazio(cuboDeDados));

		//coloca tempG no grupo vazio da estrutura principal
		preencheCuboComTempG(cuboDeDados, tempG, encontraGrupoVazio(cuboDeDados), qteElementosTempG);

		//� necessario medir o diametro dos grupos, escolher o maior e coloca-lo em grupoG
		//preenche todas as posi��es de diametroGrupo com -1
		inicializaMatrizDiametro(diametroDoGrupo);

		//preenche matriz que contem todos os diametros dos grupo existentes na estrutura principal
		preencheMatrizDeDiametro(cuboDeDados, diametroDoGrupo);


		printf("\n\n");
		for (int i = 0; i < NUM_GRUPOS; i++) {
			for (int j = 0; j < 2; j++)
				printf("%.2f\t", diametroDoGrupo[i][j]);

			printf("\n");
		}

		//encontra grupo com maior diametro
		indexMaiorDiametro = (int)identificaMaiorDiametro(diametroDoGrupo);
		//essa variavel eh importante pois sera a partir dela que preencheremosa estrutura principal
		printf("\nGrupo com maior diametro: %d", indexMaiorDiametro);

		//coloca em G o grupo com maior diametro
		//reinicializa grupoG
		inicializaMatrizNegativo(grupoG);
		//passa o grupo de maior diametro pra G
		preencheGcomMaiorDiametro(cuboDeDados, grupoG, indexMaiorDiametro);

		//atualizar qteElementos de grupoG
		qteElementos = contaElementosGrupoG(grupoG);
		fflush(stdout);

	} while (it <= maxGrupos - 2);

	//printf("\n\nQTE Elementos em G: %d\nQTE Elementos em tempG: %d\n\n", qteElementos, qteElementosTempG);

	//mostraMatrixDx(Dx);
	//mostraMatrix(grupoG);


	//printf ("\n\nGrupo vazio: %d", encontraGrupoVazio(cuboDeDados, maxGrupos));

	//antes de preencher qualquer matriz � necess�rio reseta-la, ou seja, preenche-la com numeros negativos

	//mostra a estrutura final com os grupos e seus elementos	

	for (int i = 50; i < NUM_GRUPOS; i++) {

		printf("\n\nAGt %d:\n", i);
		mostraElementosDoGrupo(cuboDeDados, i);

	}

	delta = GetTickCount() - start;
	printf("\nTempo: %.2f", delta / 1000);

	getch();
	
} //fim da int main

  //*************************************************************************************************************************************//
